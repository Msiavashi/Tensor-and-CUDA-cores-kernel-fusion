#include <hip/hip_runtime.h>
#include <stdio.h>

#define N (128 * 128)
#define ITERATIONS 100  // Increase the number of iterations

__global__ void dummy_kernel(float* data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // Dummy computation to simulate work
        float val = data[idx];
        for (int i = 0; i < 10000; i++) {  // Increase the number of iterations
            val = sinf(val);
        }
        data[idx] = val;
    }
}

int main() {
    float *d_data1, *d_data2;
    hipMalloc(&d_data1, N * sizeof(float));
    hipMalloc(&d_data2, N * sizeof(float));

    // Initialize data
    float *h_data = (float*)malloc(N * sizeof(float));
    for (int i = 0; i < N; i++) {
        h_data[i] = 1.0f;
    }
    hipMemcpy(d_data1, h_data, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_data2, h_data, N * sizeof(float), hipMemcpyHostToDevice);
    free(h_data);

    // Create two CUDA streams
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // Time measurement variables
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float parallel_time, sequential_time;

    // Test parallel execution with two streams
    hipEventRecord(start);
    for (int i = 0; i < ITERATIONS; i++) {
        dummy_kernel<<<(N+255)/256, 256, 0, stream1>>>(d_data1, N);
        dummy_kernel<<<(N+255)/256, 256, 0, stream2>>>(d_data2, N);
    }
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&parallel_time, start, stop);

    // Test sequential execution
    hipEventRecord(start);
    for (int i = 0; i < ITERATIONS; i++) {
        dummy_kernel<<<(N+255)/256, 256>>>(d_data1, N);
        dummy_kernel<<<(N+255)/256, 256>>>(d_data2, N);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&sequential_time, start, stop);

    printf("Parallel execution time (ms): %f\n", parallel_time);
    printf("Sequential execution time (ms): %f\n", sequential_time);
    printf("Speedup: %.2fx\n", sequential_time / parallel_time);

    // Cleanup
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_data1);
    hipFree(d_data2);

    return 0;
}